#include "hip/hip_runtime.h"
/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2015 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "lpbvi_cuda.h"

#include <stdio.h>

// This is not C++0x, unfortunately.
#define nullptr NULL

// This is determined by hardware, so what is below is a 'safe' guess. If this is
// off, the program might return 'nan' or 'inf'.
#define FLT_MAX 1e+35
#define FLT_MIN -1e+35

//__global__ void lpbvi_update_belief_state_dot_product_step(unsigned int n, unsigned int m, unsigned int z, unsigned int r,
//		const bool *A, const float *B, const float *T, const float *O, const float *R, float gamma,
//		const float *Gamma, const unsigned int *pi, float *GammaPrime, unsigned int *piPrime,
//		unsigned int beliefIndex, unsigned int a, unsigned int o,
//		float *maxAlphaDotBeta)
//{
//	// Each block is for a particular action, observation, and alpha-vector in Gamma_{b, a, omega} (which are Gamma^{t-1}).
//	unsigned int alphaIndex = blockIdx.x;
//
//	// Each thread is over states, and they stride if needed.
//	for (unsigned int s = threadIdx.x; s < n; s += blockDim.x) {
//		// We compute the value of this state in the alpha-vector, then multiply it by the belief, and add it to
//		// the current dot product value for this alpha-vector.
//		double value = 0.0;
//		for (unsigned int sp = 0; sp < n; sp++) {
//			value += T[s * m * n + a * n + sp] * O[a * n * z + sp * z + o] * Gamma[alphaIndex * n + sp];
//		}
//		value *= gamma * B[beliefIndex * n + s];
//		maxAlphaDotBeta[alphaIndex] += value;
//	}
//}
//
//// We do a reduction to compute the max index within maxAlphaDotBeta.
//__global__ void lpbvi_update_belief_state_max_step(unsigned int n, unsigned int m, unsigned int z, unsigned int r,
//		const bool *A, const float *B, const float *T, const float *O, const float *R, float gamma,
//		const float *Gamma, const unsigned int *pi, float *GammaPrime, unsigned int *piPrime,
//		float *belief, unsigned int action, unsigned int observation,
//		float *maxAlphaDotBeta)
//{
//
//}
//
//// We compute the alpha-vector of maxAlphaDotBeta, but instead of storing it, we add it to the alphaBAStar.
//__global__ void lpbvi_update_belief_state_max_alpha_vector_step(unsigned int n, unsigned int m, unsigned int z, unsigned int r,
//		const bool *A, const float *B, const float *T, const float *O, const float *R, float gamma,
//		const float *Gamma, const unsigned int *pi, float *GammaPrime, unsigned int *piPrime,
//		float *belief, unsigned int action, unsigned int observation, unsigned int maxAlphaIndex,
//		float *alphaBAStar)
//{
//
//}

// Find the max of each alphaBAStar using a reduction over all actions. Remember to set the action index in pi as part of this.

// For all the belief points, we execute independent code which computes the next alpha-vector to replace it (but in GammaPrime),
// using the current set of alpha-vectors (in Gamma).


__global__ void lpbvi_update(unsigned int n, unsigned int m, unsigned int z, unsigned int r,
		const bool *A, const float *B, const float *T, const float *O, const float *R, float gamma,
		const float *Gamma, const unsigned int *pi,
		float *alphaBA,
		float *GammaPrime, unsigned int *piPrime)
{
	// For each beliefIndex, we will store an alpha-vector of size n. Hence, this must be of
	// size r * n. This is used to hold intermediate values while trying to find the maximal
	// action.
//	extern __shared__ float alphaBA[];

	// Each block will run a different belief. Our overall goal: Compute the value
	// of GammaPrime[beliefIndex * n + ???] and piPrime[beliefIndex].
	unsigned int beliefIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (beliefIndex >= r) {
		return;
	}

	// Each thread deals with a different action-observation pair. Since memory can be
	// shared among threads in a block, we can easily store results in shared memory to
	// compute the maximal alpha-vector over actions.
//	unsigned int action = blockIdx.y;
//	unsigned int observation = blockIdx.z;

	// We want to compute: dot(alphaBA, belief). Instead of storing all these intermediate
	// alpha-vectors, we will just store the values of the dot products. Once we figure out
	// which action yields the largest value, then we'll compute the actual alpha-vector and
	// store it in GammaPrime[beliefIndex * n + ???], as well as set the action piPrime[beliefIndex].

	// Now we know the maximal alpha-vector for this action and observation. Compute the actual value
	// of this observation by summing each state in the alpha-vector to form a new one, plus the original
	// GammaAStar value. Since we will dot-product this with the belief, we will just do that here.

	// We want to find the action that maximizes the value, store it in piPrime, as well as its alpha-vector GammaPrime.
	float maxActionValue = FLT_MIN;

	for (unsigned int action = 0; action < m; action++) {
		// Only execute if the action is available.
		if (A[beliefIndex * m + action]) {
			// Compute Gamma_{a,*} and set it to the first value of alphaBA.
			for (unsigned int s = 0; s < n; s++) {
				alphaBA[beliefIndex * n + s] = R[s * m + action];
			}

			// Since the bottleneck is almost always read access to global memory, write access is fine here. We will
			// overwrite old alpha-vector values if this iteration is better than previous ones.
			for (unsigned int observation = 0; observation < z; observation++) {
				// Compute the max alpha vector from Gamma, given the fixed action and observation.
				float maxAlphaDotBeta = 0.0f;
				unsigned int maxAlphaIndex = 0;

				for (unsigned int alphaIndex = 0; alphaIndex < r; alphaIndex++) {
					float alphaDotBeta = 0.0f;

					for (unsigned int s = 0; s < n; s++) {
						// We compute the value of this state in the alpha-vector, then multiply it by the belief, and add it to
						// the current dot product value for this alpha-vector.
						float value = 0.0f;
						for (unsigned int sp = 0; sp < n; sp++) {
							value += T[s * m * n + action * n + sp] * O[action * n * z + sp * z + observation] * Gamma[alphaIndex * n + sp];
						}
						alphaDotBeta += gamma * value * B[beliefIndex * n + s];
					}

					// Store the maximal value and index.
					if (alphaIndex == 0 || alphaDotBeta > maxAlphaDotBeta) {
						maxAlphaDotBeta = alphaDotBeta;
						maxAlphaIndex = alphaIndex;
					}
				}

				// Now we can compute the alpha-vector component for this observation, since we have the max.
				// We will need to compute the dot product anyway, so let's just distribute the belief over the
				// sum over observations, and add it all up here.
				for (unsigned int s = 0; s < n; s++) {
					// We compute the value of this state in the alpha-vector, then multiply it by the belief, and add it to
					// the current dot product value for this alpha-vector.
					float value = 0.0f;
					for (unsigned int sp = 0; sp < n; sp++) {
						value += T[s * m * n + action * n + sp] * O[action * n * z + sp * z + observation] * Gamma[maxAlphaIndex * n + sp];
					}
					alphaBA[beliefIndex * n + s] += gamma * value;
				}
			}

			// Once the potential alpha-vector has been computed, compute the value with respect to the belief state.
			float actionValue = 0.0f;
			for (unsigned int s = 0; s < n; s++) {
				actionValue += alphaBA[beliefIndex * n + s] * B[beliefIndex * n + s];
			}

			// If this was larger, then overwrite piPrime and GammaPrime's values.
			if (actionValue > maxActionValue) {
				maxActionValue = actionValue;

				piPrime[beliefIndex] = action;
				for (unsigned int s = 0; s < n; s++) {
					GammaPrime[beliefIndex * n + s] = alphaBA[beliefIndex * n + s];
				}
			}
		}
	}
}

__global__ void lpbvi_restrict_actions(unsigned int n, unsigned int m, unsigned int z, unsigned int r,
		const float *B, const float *T, const float *O, const float *R, float eta,
		const float *Gamma, const unsigned int *pi,
		bool *A)
{
	// Each block will run a different belief. Our overall goal: Restrict the actions
	// within A[beliefIndex * n + action] for all actions a.
	unsigned int beliefIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (beliefIndex >= r) {
		return;
	}

	// First, compute the optimal value at this belief point.
	float maxAlphaDotBeta = 0.0f;

	for (unsigned int alphaIndex = 0; alphaIndex < r; alphaIndex++) {
		float alphaDotBeta = 0.0f;

		for (unsigned int s = 0; s < n; s++) {
			alphaDotBeta += Gamma[alphaIndex * n + s] * B[beliefIndex * n + s];
		}

		// Store the maximal value and index.
		if (alphaIndex == 0 || alphaDotBeta > maxAlphaDotBeta) {
			maxAlphaDotBeta = alphaDotBeta;
		}
	}

	// Assign all actions as not available.
	for (unsigned int action = 0; action < m; action++) {
		A[beliefIndex * n + action] = false;
	}

	// Now that we have the optimal value at this belief point, we can run over the
	// vectors again, and if the value at that belief state is within eta, then
	// we can mark the action in A as allowable.
	for (unsigned int alphaIndex = 0; alphaIndex < r; alphaIndex++) {
		float alphaDotBeta = 0.0f;

		for (unsigned int s = 0; s < n; s++) {
			alphaDotBeta += Gamma[alphaIndex * n + s] * B[beliefIndex * n + s];
		}

		if (maxAlphaDotBeta - alphaDotBeta < eta) {
			A[beliefIndex * n + pi[alphaIndex]] = true;
		}
	}
}

int lpbvi_cuda(unsigned int n, unsigned int m, unsigned int z, unsigned int r,
		bool *A, const float *d_B,
		const float *d_T, const float *d_O, const float *d_R,
		float gamma, float eta, unsigned int horizon,
		unsigned int numThreads,
		float *Gamma, unsigned int *pi)
{
	// The device pointers for the alpha-vectors: Gamma and GammaPrime.
	float *d_Gamma;
	float *d_GammaPrime;

	// The device pointers for the actions taken on each alpha-vector: pi and piPrime.
	unsigned int *d_pi;
	unsigned int *d_piPrime;

	// The device pointer for the intermediate alpha-vectors computed in the inner for loop.
	float *d_AlphaBA;

	// Ensure the data is valid.
	if (n == 0 || m == 0 || z == 0 || r == 0 ||
			A == nullptr || d_B == nullptr ||
			d_T == nullptr || d_O == nullptr || d_R == nullptr ||
			gamma < 0.0 || gamma >= 1.0 || horizon < 1) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s", "Invalid arguments.");
		return -1;
	}

	// Ensure threads are correct.
	if (numThreads % 32 != 0) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s", "Invalid number of threads.");
		return -2;
	}

	unsigned int numBlocks = (unsigned int)((float)r / (float)numThreads) + 1;

	// Allocate the memory on the device for A, and copy the current values.
	bool *d_A;
	if (hipMalloc(&d_A, r * m * sizeof(bool)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for the actions.");
		return -3;
	}
	if (hipMemcpy(d_A, A, r * m * sizeof(bool), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to copy memory from host to device for the actions.");
		return -3;
	}

	// Create the device-side Gamma.
	if (hipMalloc(&d_Gamma, r * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for Gamma.");
		return -3;
	}
	if (hipMemcpy(d_Gamma, Gamma, r * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to copy memory from host to device for Gamma.");
		return -3;
	}

	if (hipMalloc(&d_GammaPrime, r * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for Gamma (prime).");
		return -3;
	}
	if (hipMemcpy(d_GammaPrime, Gamma, r * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to copy memory from host to device for Gamma (prime).");
		return -3;
	}

	// Create the device-side pi.
	if (hipMalloc(&d_pi, r * sizeof(unsigned int)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for pi.");
		return -3;
	}
	if (hipMalloc(&d_piPrime, r * sizeof(unsigned int)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for pi (prime).");
		return -3;
	}

	// Create the device-side memory for the intermediate variable alphaBA.
	if (hipMalloc(&d_AlphaBA, r * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for alphaBA.");
		return -3;
	}

	// For each of the updates, run PBVI.
	for (int t = 0; t < horizon; t++) {
		fprintf(stdout, "Iteration %i of %i\n", t+1, horizon);

		// Execute a kernel for the first three stages of for-loops: B, A, Z, as a 3d-block,
		// and the 4th stage for-loop over Gamma as the threads.
		if (t % 2 == 0) {
			lpbvi_update<<< numBlocks, numThreads >>>(n, m, z, r,
					d_A, d_B, d_T, d_O, d_R, gamma,
					d_Gamma, d_pi,
					d_AlphaBA,
					d_GammaPrime, d_piPrime);
		} else {
			lpbvi_update<<< numBlocks, numThreads >>>(n, m, z, r,
					d_A, d_B, d_T, d_O, d_R, gamma,
					d_GammaPrime, d_piPrime,
					d_AlphaBA,
					d_Gamma, d_pi);
		}

		// Check if there was an error executing the kernel.
		if (hipGetLastError() != hipSuccess) {
			fprintf(stderr, "Error[lpbvi_cuda]: %s",
							"Failed to execute the 'iteration' kernel.");
			return -3;
		}

		// Wait for the kernel to finish before looping more.
		if (hipDeviceSynchronize() != hipSuccess) {
			fprintf(stderr, "Error[lpbvi_cuda]: %s",
							"Failed to synchronize the device.");
			return -3;
		}
	}
	// Copy the final result of Gamma and pi to the variables. This assumes
	// that the memory has been allocated.
	if (horizon % 2 == 1) {
		if (hipMemcpy(Gamma, d_Gamma, r * n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lpbvi_cuda]: %s",
					"Failed to copy memory from device to host for Gamma.");
			return -3;
		}
		if (hipMemcpy(pi, d_pi, r * sizeof(unsigned int), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lpbvi_cuda]: %s",
					"Failed to copy memory from device to host for pi.");
			return -3;
		}
	} else {
		if (hipMemcpy(Gamma, d_GammaPrime, r * n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lpbvi_cuda]: %s",
					"Failed to copy memory from device to host for Gamma (prime).");
			return -3;
		}
		if (hipMemcpy(pi, d_piPrime, r * sizeof(unsigned int), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lpbvi_cuda]: %s",
					"Failed to copy memory from device to host for pi (prime).");
			return -3;
		}
	}

	// Once freed, compute the available actions for the next iteration.
	lpbvi_restrict_actions<<< numBlocks, numThreads >>>(n, m, z, r,
					d_B, d_T, d_O, d_R, eta,
					d_Gamma, d_pi, d_A);

	// Check if there was an error executing the kernel.
	if (hipGetLastError() != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
						"Failed to execute the 'iteration' kernel.");
		return -3;
	}

	// Wait for the kernel to finish.
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
						"Failed to synchronize the device.");
		return -3;
	}

	// Copy the result to the r-n array A.
	if (hipMemcpy(A, d_A, r * m * sizeof(bool), hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to copy memory from device to host for the available actions at each belief state A.");
		return -3;
	}

	// Free the device-side Gamma and pi.
	hipFree(d_A);
	hipFree(d_Gamma);
	hipFree(d_GammaPrime);
	hipFree(d_pi);
	hipFree(d_piPrime);
	hipFree(d_AlphaBA);

	return 0;
}

int lpbvi_initialize_belief_points(unsigned int n, unsigned int r, const float *B, float *&d_B)
{
	// Ensure the data is valid.
	if (n == 0 || r == 0 || B == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_B, r * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_belief_points]: %s",
				"Failed to allocate device-side memory for the belief points.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_B, B, r * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_belief_points]: %s",
				"Failed to copy memory from host to device for the belief points.");
		return -3;
	}

	return 0;
}

int lpbvi_initialize_state_transitions(unsigned int n, unsigned int m, const float *T, float *&d_T)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || T == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_T, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_state_transitions]: %s",
				"Failed to allocate device-side memory for the state transitions.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_T, T, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_state_transitions]: %s",
				"Failed to copy memory from host to device for the state transitions.");
		return -3;
	}

	return 0;
}

int lpbvi_initialize_observation_transitions(unsigned int n, unsigned int m, unsigned int z,
		const float *O, float *&d_O)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || z == 0 || O == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_O, m * n * z * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_observation_transitions]: %s",
				"Failed to allocate device-side memory for the observation transitions.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_O, O, m * n * z * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_observation_transitions]: %s",
				"Failed to copy memory from host to device for the observation transitions.");
		return -3;
	}

	return 0;
}

int lpbvi_initialize_rewards(unsigned int n, unsigned int m, const float *R, float *&d_R)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || R == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_R, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_rewards]: %s",
				"Failed to allocate device-side memory for the rewards.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_R, R, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_initialize_rewards]: %s",
				"Failed to copy memory from host to device for the rewards.");
		return -3;
	}

	return 0;
}

int lpbvi_uninitialize(float *&d_B, float *&d_T, float *&d_O, float **&d_R, unsigned int k)
{
	if (d_B != nullptr) {
		hipFree(d_B);
	}
	d_B = nullptr;

	if (d_T != nullptr) {
		hipFree(d_T);
	}
	d_T = nullptr;

	if (d_O != nullptr) {
		hipFree(d_O);
	}
	d_O = nullptr;

	if (d_R != nullptr) {
		for (unsigned int i = 0; i < k; i++) {
			hipFree(d_R[i]);
		}
	}

	return 0;
}
