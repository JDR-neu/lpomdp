#include "hip/hip_runtime.h"
/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2015 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "lpbvi_cuda.h"

#include <stdio.h>

// This is not C++0x, unfortunately.
#define nullptr NULL

// This is determined by hardware, so what is below is a 'safe' guess. If this is
// off, the program might return 'nan' or 'inf'.
#define FLT_MAX 1e+35

int lpbvi_cuda(unsigned int n, unsigned int m, unsigned int z, unsigned int r,
		const bool *d_A, const float *d_B,
		const float *d_T, const float *d_R,
		float Rmin, float Rmax, float gamma, unsigned int horizon,
		unsigned int numBlocks, unsigned int numThreads,
		float *d_Gamma, unsigned int *d_pi)
{
	return 0;
}

int lpbvi_initialize_actions(unsigned int m, unsigned int r, const float *A, float *&d_A)
{
	return 0;
}

int lpbvi_initialize_belief_points(unsigned int n, unsigned int m, const float *B, float *&d_B)
{
	return 0;
}

int lpbvi_initialize_state_transitions(unsigned int n, unsigned int m, const float *T, float *&d_T)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || T == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_T, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for the state transitions.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_T, T, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to copy memory from host to device for the state transitions.");
		return -3;
	}

	return 0;
}

int lpbvi_initialize_observation_transitions(unsigned int n, unsigned int m, unsigned int z,
		const float *O, float *&d_O)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || z == 0 || O == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_O, m * n * z * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for the observation transitions.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_O, O, m * n * z * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to copy memory from host to device for the observation transitions.");
		return -3;
	}

	return 0;
}

int lpbvi_initialize_rewards(unsigned int n, unsigned int m, const float *R, float *&d_R)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || R == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_R, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to allocate device-side memory for the rewards.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_R, R, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lpbvi_cuda]: %s",
				"Failed to copy memory from host to device for the rewards.");
		return -3;
	}

	return 0;
}

int lpbvi_uninitialize(float *&d_T, float *&d_O, float *&d_R, unsigned int *&d_pi)
{
	hipFree(d_T);
	hipFree(d_O);
	hipFree(d_R);
	hipFree(d_pi);

	return 0;
}
